#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <wb.h>

#define BLOCK_SIZE 512 

#define wbCheck(stmt)                                                     \
  do {                                                                    \
    hipError_t err = stmt;                                               \
    if (err != hipSuccess) {                                             \
      wbLog(ERROR, "Failed to run stmt ", #stmt);                         \
      wbLog(ERROR, "Got CUDA error ...  ", hipGetErrorString(err));      \
      return -1;                                                          \
    }                                                                     \
  } while (0)

__global__ void scan(float* input, float* output, float* aux, int len) {
    //@@ Modify the body of this kernel to generate the scanned blocks
    //@@ Make sure to use the workefficient version of the parallel scan
    //@@ Also make sure to store the block sum to the aux array 
    __shared__ float scan_array[BLOCK_SIZE];
    int bxx = blockIdx.x * blockDim.x;
    int tx = threadIdx.x;

    if (bxx + tx < len)
        scan_array[tx] = input[bxx + tx];
    else
        scan_array[tx] = 0;

    __syncthreads();

    for (unsigned int stride = 1; stride < blockDim.x; stride *= 2)
    {
        int index = (tx + 1) * stride * 2 - 1;

        if (index < blockDim.x)
            scan_array[index] += scan_array[index - stride];
        __syncthreads();
    }

    for (unsigned int stride = BLOCK_SIZE / 4; stride > 0; stride /= 2)
    {
        int index = (tx + 1) * stride * 2 - 1;

        if (index + stride < BLOCK_SIZE)
            scan_array[index + stride] += scan_array[index];
        __syncthreads();
    }

    __syncthreads();

    if (bxx + tx < len)
        output[bxx + tx] = scan_array[tx];
    if ((aux != 0) && (tx == blockDim.x - 1))
        aux[blockIdx.x] = scan_array[tx];
}

__global__ void addScannedBlockSums(float* output, float* aux, int len) {
    //@@ Modify the body of this kernel to add scanned block sums to 
    //@@ all values of the scanned blocks
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if ((blockIdx.x != 0) && (index < len))
        output[index] += aux[blockIdx.x - 1];
    __syncthreads();
}

int main(int argc, char** argv) {
    wbArg_t args;
    float* hostInput;  // The input 1D list
    float* hostOutput; // The output 1D list
    float* deviceInput;
    float* deviceOutput;
    float* deviceAuxArray, * deviceAuxScannedArray;
    int numElements; // number of elements in the input/output list
    int numBlocks;

    args = wbArg_read(argc, argv);

    wbTime_start(Generic, "Importing data and creating memory on host");
    hostInput = (float*)wbImport(wbArg_getInputFile(args, 0), &numElements);
    hostOutput = (float*)malloc(numElements * sizeof(float));
    wbTime_stop(Generic, "Importing data and creating memory on host");

    wbLog(TRACE, "The number of input elements in the input is ",
        numElements);

    numBlocks = BLOCK_SIZE * 2;

    wbTime_start(GPU, "Allocating device memory.");
    //@@ Allocate device memory
    //you can assume that aux array size would not need to be more than BLOCK_SIZE*2 (i.e., 1024)
    hipMalloc((void**)&deviceInput, numElements * sizeof(float));
    hipMalloc((void**)&deviceOutput, numElements * sizeof(float));
    hipMalloc((void**)&deviceAuxArray, numBlocks * sizeof(float));
    hipMalloc((void**)&deviceAuxScannedArray, numBlocks * sizeof(float));
    wbTime_stop(GPU, "Allocating device memory.");

    wbTime_start(GPU, "Clearing output device memory.");
    //@@ zero out the deviceOutput using hipMemset() by uncommenting the below line
    wbCheck(hipMemset(deviceOutput, 0, numElements * sizeof(float)));
    wbTime_stop(GPU, "Clearing output device memory.");

    wbTime_start(GPU, "Copying input host memory to device.");
    //@@ Copy input host memory to device
    hipMemcpy(deviceInput, hostInput, numElements * sizeof(float), hipMemcpyHostToDevice);
    wbTime_stop(GPU, "Copying input host memory to device.");

    //@@ Initialize the grid and block dimensions here
    dim3 Grid(numBlocks, 1, 1);
    dim3 Block(BLOCK_SIZE, 1, 1);

    wbTime_start(Compute, "Performing CUDA computation");
    //@@ Modify this to complete the functionality of the scan
    //@@ on the deivce
    //@@ You need to launch scan kernel twice: 1) for generating scanned blocks 
    //@@ (hint: pass deviceAuxArray to the aux parameter)
    //@@ and 2) for generating scanned aux array that has the scanned block sums. 
    //@@ (hint: pass NULL to the aux parameter)
    //@@ Then you should call addScannedBlockSums kernel.
    scan << < Grid, Block >> > (deviceInput, deviceOutput, deviceAuxArray, numElements);
    hipDeviceSynchronize();
    scan << < Grid, Block >> > (deviceAuxArray, deviceAuxScannedArray, NULL, numBlocks);
    addScannedBlockSums << < Grid, Block >> > (deviceOutput, deviceAuxScannedArray, numElements);
    hipDeviceSynchronize();
    wbTime_stop(Compute, "Performing CUDA computation");

    wbTime_start(Copy, "Copying output device memory to host");
    //@@ Copy results from device to host	
    hipMemcpy(hostOutput, deviceOutput, numElements * sizeof(float), hipMemcpyDeviceToHost);
    wbTime_stop(Copy, "Copying output device memory to host");

    wbTime_start(GPU, "Freeing device memory");
    //@@ Deallocate device memory
    hipFree(deviceInput);
    hipFree(deviceOutput);
    hipFree(deviceAuxArray);
    hipFree(deviceAuxScannedArray);
    wbTime_stop(GPU, "Freeing device memory");

    wbSolution(args, hostOutput, numElements);

    free(hostInput);
    free(hostOutput);

    return 0;
}