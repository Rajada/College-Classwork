#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <wb.h>

#define MASK_WIDTH 5
#define MASK_RADIUS (MASK_WIDTH / 2)
#define O_TILE_WIDTH 16
#define clamp(x) (min(max((x), 0.0), 1.0))
#define BLOCK_WIDTH (O_TILE_WIDTH + (MASK_WIDTH - 1))

//@@ INSERT CODE HERE 
//implement the tiled 2D convolution kernel with adjustments for channels
//use shared memory to reduce the number of global accesses, handle the boundary conditions when loading input list elements into the shared memory
//clamp your output values

__global__ void convolutionKernel(float *P, float *N, int height, int width, int channels, const float* __restrict__ M)
{
	__shared__ float Ns[BLOCK_WIDTH][BLOCK_WIDTH];

	int tx = threadIdx.x;
	int ty = threadIdx.y;
	int outRow = blockIdx.y * O_TILE_WIDTH + ty;
	int outColumn = blockIdx.x * O_TILE_WIDTH + tx;
	int inRow = outRow - MASK_RADIUS;
	int inColumn = outColumn - MASK_RADIUS;
	float output = 0.0;

	for (int i = 0; i < channels; i++)
	{
		if ((inRow >= 0) && (inRow < height) && (inColumn >= 0) && (inColumn < width))
			Ns[ty][tx] = N[(inRow * width + inColumn) * channels + i];
		else
			Ns[ty][tx] = 0.0;

		__syncthreads();
		output = 0.0;

		if ((ty < O_TILE_WIDTH) && (tx < O_TILE_WIDTH) && (outRow < height) && (outColumn < width))
		{
			for (int j = 0; j < MASK_WIDTH; j++)
			{
				for (int k = 0; k < MASK_WIDTH; k++)
				{
					output += M[j * MASK_WIDTH + k] * Ns[j + ty][k + tx];
				}
			}

			P[(outRow * width + outColumn) * channels + i] = clamp(output);
		}
	}
}

int main(int argc, char *argv[])
{
  wbArg_t arg;
  int maskRows;
  int maskColumns;
  int imageChannels;
  int imageWidth;
  int imageHeight;
  char *inputImageFile;
  char *inputMaskFile;
  wbImage_t inputImage;
  wbImage_t outputImage;
  float *hostInputImageData;
  float *hostOutputImageData;
  float *hostMaskData;
  float *deviceInputImageData;
  float *deviceOutputImageData;
  float *deviceMaskData;

  arg = wbArg_read(argc, argv); /* parse the input arguments */

  inputImageFile = wbArg_getInputFile(arg, 0);
  inputMaskFile  = wbArg_getInputFile(arg, 1);

  inputImage   = wbImport(inputImageFile);
  hostMaskData = (float *)wbImport(inputMaskFile, &maskRows, &maskColumns);

  assert(maskRows == MASK_WIDTH);    /* mask height is fixed to 5 */
  assert(maskColumns == MASK_WIDTH); /* mask width is fixed to 5 */

  imageWidth    = wbImage_getWidth(inputImage);
  imageHeight   = wbImage_getHeight(inputImage);
  imageChannels = wbImage_getChannels(inputImage);

  outputImage = wbImage_new(imageWidth, imageHeight, imageChannels);

  hostInputImageData  = wbImage_getData(inputImage);
  hostOutputImageData = wbImage_getData(outputImage);

  wbTime_start(GPU, "Doing GPU Computation (memory + compute)");

  wbTime_start(GPU, "Doing GPU memory allocation");
  //@@ INSERT CODE HERE
  //allocate device memory
  hipMalloc((void**) &deviceInputImageData, imageWidth * imageHeight * imageChannels * sizeof(float));
  hipMalloc((void**) &deviceOutputImageData, imageWidth * imageHeight * imageChannels * sizeof(float));
  hipMalloc((void**) &deviceMaskData, maskColumns * maskRows * sizeof(float));
  wbTime_stop(GPU, "Doing GPU memory allocation");

  wbTime_start(Copy, "Copying data to the GPU");
  //@@ INSERT CODE HERE
  //copy host memory to device
  hipMemcpy(deviceInputImageData, hostInputImageData, imageWidth * imageHeight * imageChannels * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(deviceMaskData, hostMaskData, maskRows * maskColumns * sizeof(float), hipMemcpyHostToDevice);
  wbTime_stop(Copy, "Copying data to the GPU");

  wbTime_start(Compute, "Doing the computation on the GPU");
  //@@ INSERT CODE HERE
  //initialize thread block and kernel grid dimensions
  //invoke CUDA kernel
  dim3 block(BLOCK_WIDTH, BLOCK_WIDTH);
  dim3 grid((imageWidth - 1) / O_TILE_WIDTH + 1, (imageHeight - 1) / O_TILE_WIDTH + 1, 1);
  convolutionKernel <<< grid, block >>> (deviceOutputImageData, deviceInputImageData, imageHeight, imageWidth, imageChannels, deviceMaskData);
  wbTime_stop(Compute, "Doing the computation on the GPU");

  wbTime_start(Copy, "Copying data from the GPU");
  //@@ INSERT CODE HERE
  //copy results from device to host
  hipMemcpy(hostOutputImageData, deviceOutputImageData, imageWidth * imageHeight * imageChannels * sizeof(float), hipMemcpyDeviceToHost);
  wbTime_stop(Copy, "Copying data from the GPU");

  wbTime_stop(GPU, "Doing GPU Computation (memory + compute)");

  wbSolution(arg, outputImage);

  //@@ INSERT CODE HERE
  //deallocate device memory
  hipFree(deviceInputImageData);
  hipFree(deviceOutputImageData);
  hipFree(deviceMaskData);

  free(hostMaskData);
  wbImage_delete(outputImage);
  wbImage_delete(inputImage);

  return 0;
}
